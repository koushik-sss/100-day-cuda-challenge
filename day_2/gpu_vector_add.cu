#include <cstddef>
#include <hip/hip_runtime.h>
#include <iostream>

__global__ void vectorAddKernel(const float *a, const float *b, float *c,
                                int n) {

  int i = blockIdx.x * blockDim.x + threadIdx.x;

  if (i < n) {
    c[i] = a[i] + b[i];
  }
}

void vectorAddGPU(const float *host_array_a, const float *host_array_b,
                  float *host_array_c, int n) {
  size_t size = n * sizeof(float);

  float *device_array_a = nullptr;
  float *device_array_b = nullptr;
  float *device_array_c = nullptr;

  hipMalloc(&device_array_a, size);
  hipMalloc(&device_array_b, size);
  hipMalloc(&device_array_c, size);

  hipMemcpy(device_array_a, host_array_a, size, hipMemcpyHostToDevice);
  hipMemcpy(device_array_b, host_array_b, size, hipMemcpyHostToDevice);
  hipMemcpy(device_array_c, host_array_c, size, hipMemcpyHostToDevice);

  int threadsPerBlock = 512;

  int blocksPerGrid = (n + threadsPerBlock - 1) / threadsPerBlock;

  vectorAddKernel<<<blocksPerGrid, threadsPerBlock>>>(
      device_array_a, device_array_b, device_array_c, n);

  hipDeviceSynchronize();

  hipMemcpy(host_array_c, device_array_c, size, hipMemcpyDeviceToHost);

  hipFree(device_array_a);
  hipFree(device_array_b);
  hipFree(device_array_c);
}

int main() {
  const int N = 1000000;

  float *h_a = new float[N];
  float *h_b = new float[N];
  float *h_c = new float[N];

  for (int i = 0; i < N; i++) {
    h_a[i] = 1.0f;
    h_b[i] = 2.0f;
  }

  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  hipEventRecord(start);

  vectorAddGPU(h_a, h_b, h_c, N);

  hipEventRecord(stop);
  hipEventSynchronize(stop);

  float milliseconds = 0;

  hipEventElapsedTime(&milliseconds, start, stop);


  bool correct = true;
  for (int i = 0; i < N; i++) {
    if (h_c[i] != 3.0f) {
      correct = false;
      std::cout << "Error at index " << i << ": " << h_c[i] << " != 3.0"
                << std::endl;
      break;
    }
  }

  if (correct) {
    std::cout << "GPU vector addition completed successfully!" << std::endl;
  }

  std::cout << "GPU Time taken: " << milliseconds << " milliseconds"
            << std::endl;


  delete[] h_a;
  delete[] h_b;
  delete[] h_c;
  hipEventDestroy(start);
  hipEventDestroy(stop);

  return 0;
}
