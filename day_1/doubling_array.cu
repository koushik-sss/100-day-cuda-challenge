
#include <hip/hip_runtime.h>
#include <iostream>

__global__ void doubleElements(int *array, int size) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;

    if (index < size) {
        array[index] = array[index] * 2;
        
    }
}

int main() {
    const int arraySize = 15;
    int hostArray[arraySize] = {10, 20, 30, 40, 50, 60, 70, 80, 90, 100, 110, 120, 130, 140, 150};
    int* deviceArray = nullptr;
    std::cout<<"device array before hipMalloc is: "<<deviceArray<<std::endl;
    

    hipMalloc(&deviceArray, arraySize*sizeof(int));
    std::cout<<"device array after hipMalloc is: "<<deviceArray<<std::endl;


    hipMemcpy(deviceArray, hostArray, arraySize*sizeof(int), hipMemcpyHostToDevice);

    std::cout <<" Launching kernel " << std::endl;

    doubleElements<<<3,5>>>(deviceArray, arraySize);

    hipDeviceSynchronize();


    hipMemcpy(hostArray, deviceArray, arraySize*sizeof(int), hipMemcpyDeviceToHost);

    
    std::cout << "Doubled array: ";
    for (int i = 0; i < arraySize; i++) {
        std::cout << hostArray[i] << " ";
    }
    std::cout << std::endl;
    
    
    hipFree(deviceArray);
    
    return 0;

}